

#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"




int main() {

	int N[2];
	N[0]=3;N[1]=3;

	hipfftReal *d_real_data,*c_real_data;
	c_real_data = (hipfftReal*) malloc(sizeof(hipfftReal)*N[0]*N[1]);
    hipMalloc((void**)&d_real_data, N[0]*N[1] * sizeof(hipfftReal));
	c_real_data[0]=0;
	c_real_data[1]=2;
	c_real_data[2]=4;
	c_real_data[3]=6;
	c_real_data[4]=1;
	c_real_data[5]=3;
	c_real_data[6]=5;
	c_real_data[7]=7;
	c_real_data[8]=4;

	hipMemcpy(d_real_data,c_real_data,sizeof(hipfftReal)*N[0]*N[1],hipMemcpyHostToDevice);

	hipfftComplex *d_comp_data,*c_comp_data;
	hipMalloc((void**)&d_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1));
	c_comp_data = (hipfftComplex*) malloc(sizeof(hipfftComplex)*N[0]*(N[1]/2+1));

	hipfftHandle cufftForwrdHandle;
	hipfftPlan2d(&cufftForwrdHandle, N[0], N[1] , HIPFFT_R2C);
	hipfftExecR2C(cufftForwrdHandle, d_real_data, d_comp_data);
	hipDeviceSynchronize();
	hipMemcpy(c_comp_data,d_comp_data,sizeof(hipfftComplex)*N[0]*(N[1]/2+1),hipMemcpyDeviceToHost);
	hipfftDestroy(cufftForwrdHandle);

	for(int i=0;i<N[0]*(N[1]/2+1);i++)
		printf("%f %f \n",c_comp_data[i].x,c_comp_data[i].y);

// C2R

	hipfftHandle cufftInverseHandle;
	hipMemset(d_real_data,0,N[0]*N[1]*sizeof(hipfftReal));
	hipfftPlan2d(&cufftInverseHandle, N[0],N[1], HIPFFT_C2R);
	hipfftExecC2R(cufftInverseHandle, d_comp_data,d_real_data);
	hipMemcpy(c_real_data,d_real_data,sizeof(hipfftReal)*N[0]*N[1],hipMemcpyDeviceToHost);
	for(int i=0;i<N[0]*N[1];i++)
		printf("%f ",c_real_data[i]);


}
/*
ref: https://blog.csdn.net/congwulong/article/details/7576012
[0,2,4]
[6,1,3]
[5,7,4]

32 0.5+0.86i 0.5-0.86i;
-7+5.2i -1-1.73i -8.5-6.06i;
-7-5.2i -8.5+6.06i -1+1.73i;

32.000000 0.000000
0.500000 0.866025
-7.000000 5.196152
-1.000000 -1.732050
-7.000000 -5.196152
-8.500000 6.062178

inverse change :
0.000000 18.000000 36.000000 54.000000 9.000003 27.000000 45.000000 63.000000 36.000000

 */


