#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"




//#define PRINT

#define REP_TIMES 100

void testmoduleGPU(int dimx,int dimy,int dimz) {
	int N[3];
	N[0] = dimx; N[1] = dimy; N[2] = dimz;
	int LENGTH = N[0] * N[1] * N[2];
	hipfftComplex *input = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*) malloc(
			LENGTH * sizeof(hipfftComplex));
	int i;
	for (i = 0; i < N[0] * N[1]; i++) {
		input[i].x = i % 1000;
		input[i].y = 0;
	}

	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, LENGTH * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftComplex),
			hipMemcpyHostToDevice);

	hipfftHandle plan;

	int t= hipfftPlan3d(&plan, N[0], N[1], N[2], HIPFFT_C2C);


	size_t* worksize;
	worksize = (size_t*) malloc(sizeof(size_t) * 2);
	hipfftGetSize3d(plan,N[0], N[1], N[2],HIPFFT_C2C, worksize);
	printf("make : %d %ld \n",t,worksize[0]);
	hipfftExecC2C(plan, d_inputData, d_outData, HIPFFT_BACKWARD);

	hipMemcpy(output, d_outData, LENGTH * sizeof(hipfftComplex),
			hipMemcpyDeviceToHost);

	for (i = 0; i < 10; i++) {
		printf("%f %f \n", output[i].x, output[i].y);
	}

	hipfftDestroy(plan);
	free(input);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);
}

int main() {
	testmoduleGPU(200,200,200);
}
