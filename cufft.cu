#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"




//#define PRINT

#define REP_TIMES 100

float testmoduleGPU(int dimx,int dimy) {
	int N[2];
	N[0] = dimx, N[1] = dimy;
	int LENGTH = N[0] * N[1];
	hipfftComplex *input = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*) malloc(
			LENGTH * sizeof(hipfftComplex));
	int i;
	for (i = 0; i < N[0] * N[1]; i++) {
		input[i].x = i % 1000;
		input[i].y = 0;
	}

	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, N[0] * N[1] * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, N[0] * N[1] * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, input, N[0] * N[1] * sizeof(hipfftComplex),
			hipMemcpyHostToDevice);

	hipfftHandle plan;

	hipfftPlan2d(&plan, N[0], N[1], HIPFFT_C2C);

	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
	for (int i = 0; i < 100; i++) {
		//cufftExecC2C(plan, d_inputData, d_outData, CUFFT_FORWARD);
		hipfftExecC2C(plan, d_inputData, d_outData, HIPFFT_BACKWARD);
	}

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	hipMemcpy(output, d_outData, LENGTH * sizeof(hipfftComplex),
			hipMemcpyDeviceToHost);
/*
	for (i = 0; i < 10; i++) {
		printf("%f %f \n", output[i].x, output[i].y);
	}
	*/
	hipfftDestroy(plan);
	free(input);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);
	return msecTotal1;
}

int main() {
	double timeres[10];
    //128=2^7    ; 8192=2^13
	int pownum=3;
	for(pownum=7;pownum<=13;pownum++)
	{
		double avertime = 0;
		for (int i = 0; i < 10; i++) {
			timeres[i] = testmoduleGPU(pow(2,pownum),pow(2,pownum));
			printf("ITER %f ", timeres[i]);
			avertime += timeres[i];
		}
		printf("\n AVER %f \n", avertime / 10);
	}
}
