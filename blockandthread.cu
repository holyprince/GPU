#include "hip/hip_runtime.h"
int nx = 1 << 14;  
int ny = 1 << 14;  
int dimx = 32;  
dim3 block(dimx, 1);  
dim3 grid((nx + block.x - 1) / block.x, 1); 


__global__ void kernel_function()  
{  
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;  
  
    if (ix < nx )  
    {  
        do_something();  
    }  
}  


int nx = 1 << 14;  
int ny = 1 << 14;  
int dimx = 32;  
dim3 block(dimx, 1);  
dim3 grid((nx + block.x - 1) / block.x, ny); 


__global__ void kernel_function()  
{  
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;  
    unsigned int iy = blockIdx.y;  
    unsigned int idx = iy * nx + ix;  
  
    if (ix < nx && iy < ny)  
    {  
        do_something();  
    }  
}  


int nx = 1 << 14;  
int ny = 1 << 14;  
int dimx = 32;  
int dimy = 32;  
dim3 block(dimx, dimy);  
dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);  

__global__ void kernel_function()  
{  
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;  
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;  
    unsigned int idx = iy * nx + ix;  
  
    if (ix < nx && iy < ny)  
    {  
        do_something();  
    }  
}