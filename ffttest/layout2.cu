#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"


#define NDIM 3
#define NX 3
#define NY 3
#define NZ 4


void layoutxyz(hipfftComplex *data,hipfftComplex *data2)
{
	int rawX=(NX/2+1);
	for(int z=0;z<NZ;z++)
		for(int y=0;y<NY;y++)
			for(int x=0;x<(NX/2+1);x++)
			{
				int desx,desy,desz;
				data2[z*NX*NY+y*NX+x].x=data[z*rawX*NY+y*rawX+x].x;
				data2[z*NX*NY+y*NX+x].y=data[z*rawX*NY+y*rawX+x].y;
				if(x==0)
					desx=0;
				else
					desx=NX-x;
				if (desx >= rawX) {
					if (y == 0)
						desy = 0;
					else
						desy = NY - y;
					data2[z*NX*NY+desy*NX+desx].x = data2[z*NX*NY+y*NX+x].x;
					data2[z*NX*NY+desy*NX+desx].y = -data2[z*NX*NY+y*NX+x].y;
				}
			}
}

void layoutxy(int *data,int *data2)
{
	int rawX=(NX/2+1);

		for(int y=0;y<NY;y++)
			for(int x=0;x<(NX/2+1);x++)
			{
				int dexx,desy,desz;
				if(x==0)
					dexx=0;
				else
					dexx=NX-x;
				if (y == 0)
					desy = 0;
				else
					desy = NY - y;
				if (dexx >= rawX) {
					data2[desy * NX + dexx] = -data[y * rawX + x];
					data2[y * NX + x] = data[y * rawX + x];
				}
				else
					data2[desy * NX + dexx] = data[y * rawX + x];
			}
}

void layout(int *data)
{
	int rawX=(NX/2+1);
// from back to start
	for(int i=(rawX*NY-1);i>=0;i--)
	{
		int y=i/NY;
		int x=i%rawX;
		data[y * rawX + x] = data[i];
	}
/*		for(int y=0;y<NY;y++)
			for(int x=0;x<(NX/2+1);x++)
			{
				data[y * NX + x] = data[y * rawX + x];
			}*/
	/*	for(int y=0;y<NY;y++)
			for(int x=(NX/2+1);x<NX;x++)
			{

				data[NY - y * NX + NX-x] = data[y * NX + x];
			}*/
}
int main()
{


	int N[3];
	N[0] = NX; N[1] = NY; N[2] = NZ;
	int LENGTH = N[0] * N[1] * N[2];

	hipfftReal *input = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));
	hipfftComplex *output_data = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data2 = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftReal *outputreal = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));
	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i] = i * i +i ;
	}

	hipfftReal *d_inputData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftReal));
	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftReal),hipMemcpyHostToDevice);

	hipfftComplex *d_output ;
	hipMalloc((void**) &d_output, LENGTH * sizeof(hipfftComplex));
	hipfftComplex *d_output2 ;
	hipMalloc((void**) &d_output2, LENGTH * sizeof(hipfftComplex));

	hipfftHandle plan1,plan2,plan3;

	int t= hipfftPlan3d(&plan1, N[0], N[1], N[2], HIPFFT_R2C);


	//cufftGetSize3d(plan,N[0], N[1], N[2],CUFFT_C2C, worksize);
	//printf("make : %d %ld \n",t,worksize[0]);
	hipfftExecR2C(plan1, d_inputData, d_output);

	hipMemcpy(output_data, d_output, (N[0]/2+1)*N[1]*N[2] * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (i = 0; i < (N[0]/2+1)*N[1]*N[2]; i++) {
		printf("%f %f \n", output_data[i].x, output_data[i].y);
	}
	layoutxyz(output_data,output_data2);
	printf("=========================\n");
	for (i = 0; i < (N[0])*N[1]*N[2]; i++) {
		printf("%f %f \n", output_data2[i].x, output_data2[i].y);
	}

	t= hipfftPlan3d(&plan3, N[0], N[1], N[2], HIPFFT_C2R);


	//cufftGetSize3d(plan,N[0], N[1], N[2],CUFFT_C2C, worksize);
	//printf("make : %d %ld \n",t,worksize[0]);
	hipfftExecC2R(plan3, d_output, d_inputData);
	hipMemcpy(input, d_inputData, LENGTH * sizeof(hipfftReal),hipMemcpyDeviceToHost);
	printf("=====normal C2R =========\n");
	for (i = 0; i < (N[0])*N[1]*N[2]; i++) {
		printf("%f  \n",input[i]/(N[0]*N[1]*N[2]));
	}
	//C2C

	hipMemcpy(d_output2, output_data2, (N[0])*N[1]*N[2] * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	t= hipfftPlan3d(&plan2, N[0], N[1], N[2], HIPFFT_C2C);
	hipfftExecC2C(plan2, d_output2, d_output2,HIPFFT_BACKWARD);
	hipMemcpy(output_data2,d_output2 , (N[0])*N[1]*N[2] * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	printf("=========================\n");
	for (i = 0; i < (N[0])*N[1]*N[2]; i++) {
		printf("%f %f \n", output_data2[i].x/(N[0]*N[1]*N[2]), output_data2[i].y);
	}

/*
	layout(output_data);

	printf("After :\n");
	cudaMemcpy(d_output, output_data, LENGTH * sizeof(cufftComplex), cudaMemcpyHostToDevice);

	cufftPlan3d(&plan2, N[0], N[1], N[2], CUFFT_C2C);
	cufftExecC2C(plan2, d_output, d_output,CUFFT_INVERSE);
	cudaMemcpy(output_data2, d_output, LENGTH * sizeof(cufftReal), cudaMemcpyDeviceToHost);
	for (i = 0; i < 10; i++) {
		printf("%f \n", output_data2[i].x/LENGTH);
	}





	cufftDestroy(plan1);
	cufftDestroy(plan2);
	free(input);
	free(output_data);
	cudaFree(d_inputData);
	cudaFree(d_output);*/
}
