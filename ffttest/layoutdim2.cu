#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"


#define NDIM 3
#define NX 3
#define NY 4
#define NZ 4

/*
void layoutxyz(cufftComplex *data,cufftComplex *data2)
{
	int rawX=(NX/2+1);
	for(int z=0;z<NZ;z++)
		for(int y=0;y<NY;y++)
			for(int x=0;x<(NX/2+1);x++)
			{
				int desx,desy,desz;
				data2[z*NX*NY+y*NX+x].x=data[z*rawX*NY+y*rawX+x].x;
				data2[z*NX*NY+y*NX+x].y=data[z*rawX*NY+y*rawX+x].y;
				if(x==0)
					desx=0;
				else
					desx=NX-x;
				if (desx >= rawX) {
					if (y == 0)
						desy = 0;
					else
						desy = NY - y;
					data2[z*NX*NY+desy*NX+desx].x = data2[z*NX*NY+y*NX+x].x;
					data2[z*NX*NY+desy*NX+desx].y = -data2[z*NX*NY+y*NX+x].y;
				}
			}
}*/
void layoutxy(hipfftComplex *data,hipfftComplex *data2)
{
	int rawY= NY/2+1;

	for (int x = 0; x < NX; x++)
		for (int y = 0; y < rawY; y++) {

			data2[x*NY+y].x=data[x*rawY+y].x;
			data2[x*NY+y].y=data[x*rawY+y].y;
		}
	for (int x = 0; x < NX; x++)
		for (int y = rawY; y < NY; y++) {
			int desx,desy;
			if (x == 0)
				desx = 0;
			else
				desx = NX - x;
			desy=NY - y;

			data2[x * NY + y].x=data2[desx * NY + desy].x;
			data2[x * NY + y].y= - data2[desx * NY + desy].y;
		}


}



int main()
{


	int N[2];
	N[0] = NX; N[1] = NY;
	int LENGTH = N[0] * N[1] ;

	hipfftReal *input = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));
	hipfftComplex *inputcccc = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data2 = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftReal *outputreal = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));

	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i] = i * i +i ;
		inputcccc[i].x=i * i +i ;
		inputcccc[i].y=0;
	}



	hipfftReal *d_inputData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftReal));
	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftReal),hipMemcpyHostToDevice);

	hipfftComplex *d_inputCom;
	hipMalloc((void**) &d_inputCom, LENGTH * sizeof(hipfftComplex));
	hipMemcpy(d_inputCom, inputcccc, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);


	hipfftComplex *d_output ;
	hipMalloc((void**) &d_output, LENGTH * sizeof(hipfftComplex));
	hipfftComplex *d_output2 ;
	hipMalloc((void**) &d_output2, LENGTH * sizeof(hipfftComplex));

	hipfftHandle plan1,plan2,plan3,plan4;

	int t= hipfftPlan2d(&plan1, N[0], N[1], HIPFFT_R2C);
	t= hipfftPlan2d(&plan4, N[0], N[1], HIPFFT_C2C);


	hipfftExecR2C(plan1, d_inputData, d_output);



	hipMemcpy(output_data, d_output, N[0]*(N[1]/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	printf("=======R2C========\n");
	for (i = 0; i < N[0]*(N[1]/2+1); i++) {
		printf("%f %f \n", output_data[i].x, output_data[i].y);
	}

	memset(output_data2,0,LENGTH*sizeof(hipfftComplex));
	layoutxy(output_data,output_data2);
	printf("======after layout ====\n");
	for (i = 0; i < LENGTH; i++) {
		printf("%f %f \n", output_data2[i].x, output_data2[i].y);
	}


	hipMemcpy(d_output2, output_data2, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	t= hipfftPlan2d(&plan2, N[0], N[1], HIPFFT_C2C);
	hipfftExecC2C(plan2, d_output2, d_output2,HIPFFT_BACKWARD);
	hipMemcpy(output_data2,d_output2 , LENGTH* sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	printf("=========================\n");
	for (i = 0; i < LENGTH; i++) {
		printf("%f %f \n", output_data2[i].x/(LENGTH), output_data2[i].y);
	}



}