#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include ""



void changehertonature(hipfftComplex *data)
{
	//for(int z=0;z<NZ;z++)
		for(int y=0;y<NY;y++)
			for(int x=0;x<(NX/2+1);x++)
			{
				int dexx,desy,desz;
				if(x==0)
					dexx=0;
				else
					dexx=NX-x;
				if(y==0)
					desy=0;
				else
					desy=NY-y;
				data[desy*NX+dexx].x=data[y*NX+x].x;
				data[desy*NX+dexx].y=data[y*NX+x].y;
				     [dexx][desy].x=data[x][y][z].x;
				data[dexx][desy].y=-data[x][y][z].y;
			}
}

int main() {

	int N[2];
	N[0]=4;N[1]=4;

	hipfftReal *d_real_data,*c_real_data;
	c_real_data = (hipfftReal*) malloc(sizeof(hipfftReal)*N[0]*N[1]);
    hipMalloc((void**)&d_real_data, N[0]*N[1] * sizeof(hipfftReal));
	c_real_data[0]=0;
	c_real_data[1]=2;
	c_real_data[2]=4;
	c_real_data[3]=6;
	c_real_data[4]=1;
	c_real_data[5]=3;
	c_real_data[6]=5;
	c_real_data[7]=7;
	c_real_data[8]=4;
	c_real_data[9]=4;
	c_real_data[10]=4;
	c_real_data[11]=4;
	c_real_data[1]=4;

	hipMemcpy(d_real_data,c_real_data,sizeof(hipfftReal)*N[0]*N[1],hipMemcpyHostToDevice);

	hipfftComplex *d_comp_data,*c_comp_data;
	hipMalloc((void**)&d_comp_data, sizeof(hipfftComplex)*N[0]*(N[1]/2+1));
	c_comp_data = (hipfftComplex*) malloc(sizeof(hipfftComplex)*N[0]*(N[1]/2+1));

	hipfftHandle cufftForwrdHandle;
	hipfftPlan2d(&cufftForwrdHandle, N[0], N[1] , HIPFFT_R2C);
	hipfftExecR2C(cufftForwrdHandle, d_real_data, d_comp_data);
	hipDeviceSynchronize();
	hipMemcpy(c_comp_data,d_comp_data,sizeof(hipfftComplex)*N[0]*(N[1]/2+1),hipMemcpyDeviceToHost);
	hipfftDestroy(cufftForwrdHandle);

	for(int i=0;i<N[0]*(N[1]/2+1);i++)
		printf("%f %f \n",c_comp_data[i].x,c_comp_data[i].y);

// C2R

	hipfftHandle cufftInverseHandle;
	hipMemset(d_real_data,0,N[0]*N[1]*sizeof(hipfftReal));
	hipfftPlan2d(&cufftInverseHandle, N[0],N[1], HIPFFT_C2R);
	hipfftExecC2R(cufftInverseHandle, d_comp_data,d_real_data);
	hipMemcpy(c_real_data,d_real_data,sizeof(hipfftReal)*N[0]*N[1],hipMemcpyDeviceToHost);
	for(int i=0;i<N[0]*N[1];i++)
		printf("%f ",c_real_data[i]);

//C2C
	hipfftComplex *d_in_data,*c_in_data;
	hipMalloc((void**)&d_in_data, sizeof(hipfftComplex)*N[0]*N[1]);
	c_in_data = (hipfftComplex*) malloc(sizeof(hipfftComplex)*N[0]*N[1]);

	c_in_data[0].x=0;	c_in_data[0].y=0;
	c_in_data[1].x=2;	c_in_data[1].y=0;
	c_in_data[2].x=4;	c_in_data[2].y=0;
	c_in_data[3].x=6;	c_in_data[3].y=0;
	c_in_data[4].x=1;	c_in_data[4].y=0;
	c_in_data[5].x=3;	c_in_data[5].y=0;
	c_in_data[6].x=5;	c_in_data[6].y=0;
	c_in_data[7].x=7;	c_in_data[7].y=0;
	c_in_data[8].x=4;	c_in_data[8].y=0;

	hipMemcpy(d_in_data,c_in_data,sizeof(hipfftComplex)*N[0]*N[1],hipMemcpyHostToDevice);

	hipfftHandle CCForwrdHandle;
	hipfftPlan2d(&CCForwrdHandle, N[0], N[1] , HIPFFT_C2C);
	hipfftExecC2C(CCForwrdHandle, d_in_data, d_in_data,HIPFFT_FORWARD);
	hipDeviceSynchronize();
	hipMemcpy(c_in_data,d_in_data,sizeof(hipfftComplex)*N[0]*N[1],hipMemcpyDeviceToHost);
	hipfftDestroy(CCForwrdHandle);

	printf("\n");
	for(int i=0;i<N[0]*N[1];i++)
		printf("%f %f \n",c_in_data[i].x,c_in_data[i].y);




}
