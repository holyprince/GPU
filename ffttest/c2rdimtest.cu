
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipfft/hipfft.h"

#define NX 3
#define NY 3
#define LENGTH NX*NY
#define LENGTHC (NX/2+1)*NY
int main()
{



	float data[9];  //,data2[9];
	int i;
	//data[0]=1;data[1]=2;data[2]=3;data[3]=4;data[4]=-3;data[5]=6;

	for(int i=0;i<NX*NY;i++)
	{
		data[i]=i+2;
		printf("%f ",data[i]);
		if(i%3==2)
			printf("\n");
	}

	hipfftComplex *output_data = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftReal *output_datareal = (hipfftReal*) malloc( LENGTH * sizeof(hipfftReal));

	hipfftReal *d_inputData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftReal));
	hipMemcpy(d_inputData, data, LENGTH * sizeof(hipfftReal),hipMemcpyHostToDevice);

	hipfftComplex *d_output ;
	hipMalloc((void**) &d_output, LENGTH * sizeof(hipfftComplex));

/*	cufftHandle plan1,planc2r;

	int t= cufftPlan2d(&plan1, NX, NY, CUFFT_R2C);
	cufftExecR2C(plan1, d_inputData, d_output);

	cudaMemcpy(output_data, d_output, (NX/2+1)*NY * sizeof(cufftComplex), cudaMemcpyDeviceToHost);

	for (i = 0; i < (NX/2+1)*NY; i++) {
		printf("%f %f\n", output_data[i].x, output_data[i].y);
	}

	t= cufftPlan2d(&planc2r, NX, NY, CUFFT_C2R);
	cufftExecC2R(planc2r, d_output, (cufftReal *)d_output);

	cudaMemcpy(output_datareal, d_output, NX*NY * sizeof(cufftReal), cudaMemcpyDeviceToHost);

	for (i = 0; i < NX*NY; i++) {
		printf("%f \n", output_datareal[i]/(NX*NY));
	}
*/
	hipfftHandle plan1,plan2;
	size_t workSize[2];

/*
	int rank=1;
	int n[1];
	n[0]=NX;
	int istride=1;
	int idist = NX;
	int ostride=1;
	int odist = NX;
	int inembed[2];
	int onembed[2];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[0] = NY;
	cufftPlanMany(&plan1,rank,n,inembed, istride ,idist , onembed, ostride,odist, CUFFT_R2C, NY);
	cufftExecR2C(plan1, d_inputData, d_output);
	rank=1;
	n[0]=NY;
	istride=NX;
	idist = 1;
	ostride=NX;
	odist = 1;
	inembed[2];
	onembed[2];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[0] = NY;

	cufftPlanMany(&plan2,rank,n,inembed, istride ,idist , onembed, ostride,odist, CUFFT_C2C, NX);

	cufftExecC2C(plan2, d_output, d_output,CUFFT_FORWARD);

	cudaMemcpy(output_data, d_output, NX*NY * sizeof(cufftComplex), cudaMemcpyDeviceToHost);
	for (i = 0; i < NX*NY; i++) {
		printf("%f %f\n", output_data[i].x, output_data[i].y);
	}
*/

	int rank=1;
	int n[1];
	n[0]=NY;
	int istride=NX;
	int idist = 1;
	int ostride=NX;
	int odist = 1;
	int inembed[2];
	int onembed[2];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[0] = NY;
	hipfftPlanMany(&plan2,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_R2C, NX);

	hipfftExecR2C(plan2, d_inputData, d_output);



	rank=1;

	n[0]=NX;
	istride=1;
	idist = NX;
	ostride=1;
	odist = NX;
	inembed[2];
	onembed[2];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[0] = NY;
	hipfftPlanMany(&plan1,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_C2C, NY);
	hipfftExecC2C(plan1, d_output, d_output,HIPFFT_FORWARD);




	hipMemcpy(output_data, d_output, NX*NY * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for (i = 0; i < NX*NY; i++) {
		printf("%f %f\n", output_data[i].x, output_data[i].y);
	}



	printf("data: \n%f  \n",(data+3)[0]);

	return 0;
}

/*
 *
9.000000 -0.000000
-1.500000 0.866025
18.000000 0.000000
-1.499999 0.866025
27.000000 -0.000000
-1.499999 0.866025
 */
