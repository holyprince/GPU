#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"


#define NDIM 3
#define NX 3
#define NY 4
#define NZ 4

/*
void layoutxyz(cufftComplex *data,cufftComplex *data2)
{
	int rawX=(NX/2+1);
	for(int z=0;z<NZ;z++)
		for(int y=0;y<NY;y++)
			for(int x=0;x<(NX/2+1);x++)
			{
				int desx,desy,desz;
				data2[z*NX*NY+y*NX+x].x=data[z*rawX*NY+y*rawX+x].x;
				data2[z*NX*NY+y*NX+x].y=data[z*rawX*NY+y*rawX+x].y;
				if(x==0)
					desx=0;
				else
					desx=NX-x;
				if (desx >= rawX) {
					if (y == 0)
						desy = 0;
					else
						desy = NY - y;
					data2[z*NX*NY+desy*NX+desx].x = data2[z*NX*NY+y*NX+x].x;
					data2[z*NX*NY+desy*NX+desx].y = -data2[z*NX*NY+y*NX+x].y;
				}
			}
}*/
void layoutxy(hipfftComplex *data,hipfftComplex *data2)
{
	int rawY= NY/2+1;

	for (int x = 0; x < NX; x++)
		for (int y = 0; y < rawY; y++) {

			data2[x*NY+y].x=data[x*rawY+y].x;
			data2[x*NY+y].y=data[x*rawY+y].y;
		}
	for (int x = 0; x < NX; x++)
		for (int y = rawY; y < NY; y++) {
			int desx,desy;
			if (x == 0)
				desx = 0;
			else
				desx = NX - x;
			desy=NY - y;

			data2[x * NY + y].x=data2[desx * NY + desy].x;
			data2[x * NY + y].y= - data2[desx * NY + desy].y;
		}


}



int main()
{


	int N[2];
	N[0] = NX; N[1] = NY;
	int LENGTH = N[0] * N[1] ;

	hipfftReal *input = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));
	hipfftComplex *inputcccc = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data2 = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftReal *outputreal = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));

	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i] = i * i +i ;
		inputcccc[i].x=i * i +i ;
		inputcccc[i].y=0;
		printf("%f ",input[i]);
		if((i+1)%NY==0)
			printf("\n");
	}



	hipfftReal *d_inputData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftReal));
	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftReal),hipMemcpyHostToDevice);

	hipfftComplex *d_inputCom;
	hipMalloc((void**) &d_inputCom, LENGTH * sizeof(hipfftComplex));
	hipMemcpy(d_inputCom, inputcccc, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);


	hipfftComplex *d_output ;
	hipMalloc((void**) &d_output, N[0]*(N[1]/2+1)  * sizeof(hipfftComplex));
	hipfftComplex *d_output2 ;
	hipMalloc((void**) &d_output2, LENGTH * sizeof(hipfftComplex));

	hipfftHandle plan1,plan2,plan3,plan4;

	int t= hipfftPlan2d(&plan1, N[0], N[1], HIPFFT_R2C);
	t= hipfftPlan2d(&plan4, N[0], N[1], HIPFFT_C2C);


	hipfftExecR2C(plan1, d_inputData, d_output);



	hipMemcpy(output_data, d_output, N[0]*(N[1]/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	printf("=======R2C========\n");
	for (i = 0; i < N[0]*(N[1]/2+1); i++) {
		printf("%f=%f ", output_data[i].x, output_data[i].y);
		if((i+1)%(NY/2+1)==0)
			printf("\n");
	}

	memset(output_data2,0,LENGTH*sizeof(hipfftComplex));
	layoutxy(output_data,output_data2);
	printf("======after layout ====\n");
	for (i = 0; i < LENGTH; i++) {
		printf("%f=%f ", output_data2[i].x, output_data2[i].y);
		if((i+1)%NY==0)
			printf("\n");
	}


	hipMemcpy(d_output2, output_data2, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	t= hipfftPlan2d(&plan2, N[0], N[1], HIPFFT_C2C);
	hipfftExecC2C(plan2, d_output2, d_output2,HIPFFT_BACKWARD);
	hipMemcpy(output_data2,d_output2 , LENGTH* sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	printf("=========================\n");
	for (i = 0; i < LENGTH; i++) {
		printf("%f=%f ", output_data2[i].x/(LENGTH), output_data2[i].y);
		if((i+1)%NY==0)
			printf("\n");
	}

/*
	0.000000 2.000000 6.000000 12.000000
	20.000000 30.000000 42.000000 56.000000
	72.000000 90.000000 110.000000 132.000000
	=======R2C========
	572.000000=0.000002 -66.000000=78.000000 -72.000000=-0.000002
	-255.999985=221.702499 10.143601=-37.856403 23.999985=-13.856392
	-256.000000=-221.702515 37.856411=-10.143597 24.000000=13.856415
	======after layout ====
	572.000000=0.000002 -66.000000=78.000000 -72.000000=-0.000002 -66.000000=-78.000000
	-255.999985=221.702499 10.143601=-37.856403 23.999985=-13.856392 37.856411=10.143597
	-256.000000=-221.702515 37.856411=-10.143597 24.000000=13.856415 10.143601=37.856403
	=========================
	0.000001=0.000008 2.000001=-0.000034 5.999997=0.000008 12.000001=-0.000034
	19.999998=-0.000004 30.000000=0.000049 42.000004=-0.000004 56.000000=0.000049
	72.000000=-0.000004 90.000000=-0.000004 110.000000=-0.000004 132.000000=-0.000004
*/
	// raw data R2C normal should C2R
/*
	cufftHandle planc2r;

	//t= cufftPlan2d(&planc2r, N[0], N[1], CUFFT_C2R);

	if (cufftPlanMany(&planc2r, 2, N,
			NULL, 1, 0,
			NULL, 1, 0,
			CUFFT_C2R, 1) != CUFFT_SUCCESS);


	cufftExecC2R(planc2r, d_output,(cufftReal *)d_output);
	cudaDeviceSynchronize();
	cudaMemcpy(outputreal, d_output, N[0]*(N[1]/2+1)  * sizeof(cufftComplex), cudaMemcpyDeviceToHost);

	printf("==========c2r===========\n");
	int linenum = (N[1]/2+1) *2;
	for (i = 0; i < N[0]*(N[1]/2+1) *2; i++) {
		printf("%f ", outputreal[i]/(LENGTH));
		if((i+1)%linenum==0)
			printf("\n");
	}
*/
	// first do C2C inverse
	hipfftHandle plandiv;
	hipfftPlan1d(&plandiv, N[1], HIPFFT_C2C,N[0]);
	hipfftExecC2C(plandiv, d_output, d_output,HIPFFT_BACKWARD);
	// then C2R inverse
	int dim[1]; dim[0]=N[0];
	int inembed[2];
	int outembed[2];
	inembed[0]=N[0];
	inembed[1]=N[1];
	outembed[0]=N[0];
	outembed[1]=N[1];
	int stride= N[1];
	int distance= 1;
	hipfftHandle plandivcr;
	hipfftPlanMany(&plandivcr, 1, dim, inembed, stride, distance, outembed, stride, distance,HIPFFT_C2R, N[1]);
	hipfftExecC2R(plandivcr, d_output,(hipfftReal *)d_output);
	hipDeviceSynchronize();
	hipMemcpy(outputreal, d_output, N[0]*(N[1]/2+1)  * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	printf("==========c2r===========\n");
	int linenum = (N[1]/2+1) *2;
	for (i = 0; i < N[0]*(N[1]/2+1) *2; i++) {
		printf("%f ", outputreal[i]/(LENGTH));
		if((i+1)%linenum==0)
			printf("\n");
	}


}
