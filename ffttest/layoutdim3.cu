#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"


#define NDIM 3
#define NX 3
#define NY 4
#define NZ 5
#define COMLEN N[2]*N[0]*(N[1]/2+1)


void layoutxyz(hipfftComplex *data,hipfftComplex *data2)
{
	int rawY= NY/2+1;


	for(int z=0;z< NZ;z++)
		for (int x = 0; x < NX; x++)
		{
			memcpy(data2+z*NX*NY+x*NY,data+z*NX*rawY+x*rawY,rawY*sizeof(hipfftComplex));
		}
/*	for(int z=0;z< NZ;z++)
	for (int x = 0; x < NX; x++)
		for (int y = 0; y < rawY; y++) {

			data2[z*NX*NY+x*NY+y].x=data[z*NX*rawY+x*rawY+y].x;
			data2[z*NX*NY+x*NY+y].y=data[z*NX*rawY+x*rawY+y].y;
		}*/
	for(int z=0;z< NZ;z++)
	for (int x = 0; x < NX; x++)
		for (int y = rawY; y < NY; y++) {
			int desx,desy,desz;
			if (x == 0)
				desx = 0;
			else
				desx = NX - x;
			if(z==0)
				desz =0;
			else
				desz = NZ-z;
			desy=NY - y;

			data2[z*NX*NY+x * NY + y].x=data2[desz*NX*NY+desx * NY + desy].x;
			data2[z*NX*NY+x * NY + y].y= - data2[desz*NX*NY+desx * NY + desy].y;
		}


}



int main()
{


	int N[3];
	N[0] = NX; N[1] = NY; N[2] = NZ;
	int LENGTH = N[0] * N[1] *N[2];
	hipfftReal *input = (hipfftReal*) malloc(LENGTH * sizeof(hipfftReal));
	hipfftComplex *inputcccc = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output_data2 = (hipfftComplex*) malloc( LENGTH * sizeof(hipfftComplex));
	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i] = i * i +i ;
		inputcccc[i].x= i * i +i ;
		inputcccc[i].y=0;
	}


	hipfftComplex *d_inputCom;
	hipMalloc((void**) &d_inputCom, LENGTH * sizeof(hipfftComplex));
	hipMemcpy(d_inputCom, inputcccc, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);


	hipfftComplex *d_output ;
	hipMalloc((void**) &d_output, LENGTH * sizeof(hipfftComplex));


	hipfftReal *d_inputData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftReal));
	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftReal),hipMemcpyHostToDevice);

	hipfftHandle plan1,plan2;

	hipfftPlan3d(&plan1, N[2], N[0], N[1], HIPFFT_R2C);
	hipfftPlan3d(&plan2, N[2], N[0], N[1], HIPFFT_C2C);
	hipfftExecR2C(plan1, d_inputData, d_output);

	hipMemcpy(output_data, d_output, COMLEN * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for (i = 0; i < COMLEN; i++) {
		printf("%f %f \n", output_data[i].x, output_data[i].y);
	}
	layoutxyz(output_data,output_data2);
	printf("=======output_data2========\n");
	for (i = 0; i < LENGTH; i++) {
		printf("%f %f \n", output_data2[i].x, output_data2[i].y);
	}

	hipMemcpy(d_output, output_data2, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftExecC2C(plan2, d_output, d_output,HIPFFT_BACKWARD);
	hipMemcpy(output_data, d_output, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	printf("=======C2C========\n");
	for (i = 0; i < LENGTH; i++) {
		printf("%f %f \n", output_data[i].x/LENGTH, output_data[i].y);
	}

}