#include "hip/hip_runtime.h"

// System includes
#include <stdlib.h>
#include <stdio.h>

#include <string.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

//CUFFT Header file
#include <hipfft/hipfftXt.h>
#include "timer.h"

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

// Complex data type
typedef float2 Complex;


#define N 512



///////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{

	int threadsPerBlock;
	int blocksPerGrid ;
	int GPU_N;
	(hipGetDeviceCount(&GPU_N));
	int nGPUs = 2;
	int *whichGPUs;
	whichGPUs = (int*) malloc(sizeof(int) * nGPUs);

	Complex *f = (Complex*) malloc(sizeof(Complex) * N * N *N);
	float *u_a = (float*) malloc(sizeof(float) * N * N*N);
	Complex *h_d_out = (Complex *) malloc(sizeof(Complex) * N * N*N);

	for (int i = 0; i < N * N*N; i++) {
		f[i].x = i % 5000 ;
		f[i].y = 0;
	}
	StartTimer();
	// hipfftCreate() - Create an empty plan
	hipfftResult result;
	hipfftHandle planComplex;
	result = hipfftCreate(&planComplex);
	if (result != HIPFFT_SUCCESS) {
		printf("hipfftCreate failed\n");
		exit(EXIT_FAILURE);
	}

	// hipfftXtSetGPUs() - Define which GPUs to use
	result = hipfftXtSetGPUs(planComplex, nGPUs, whichGPUs);

	if (result == HIPFFT_INVALID_DEVICE) {
		printf("This sample requires two GPUs on the same board.\n");
		printf("No such board was found. Waiving sample.\n");

	} else if (result != HIPFFT_SUCCESS) {
		printf("hipfftXtSetGPUs failed\n");

	}

	//Print the device information to run the code
	printf("\nRunning on GPUs\n");

	size_t* worksize;
	worksize = (size_t*) malloc(sizeof(size_t) * nGPUs);

	// hipfftMakePlan2d() - Create the plan
	result = hipfftMakePlan3d(planComplex, N, N,N, HIPFFT_C2C, worksize);

	if (result != HIPFFT_SUCCESS) {
		printf("*MakePlan* failed\n");
		exit(EXIT_FAILURE);
	}

	// Create a variable on device
	// d_f - variable on device to store the input data
	// d_d_f - variable that store the natural order of d_f data
	// d_out - device output
	hipLibXtDesc *d_f, *d_d_f;

	result = hipfftXtMalloc(planComplex, (hipLibXtDesc **) &d_f, HIPFFT_XT_FORMAT_INPLACE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMalloc failed\n");
		exit(EXIT_FAILURE);
	}
	result = hipfftXtMalloc(planComplex, (hipLibXtDesc **) &d_d_f, HIPFFT_XT_FORMAT_INPLACE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMalloc failed\n");
		exit(EXIT_FAILURE);
	}

	// hipfftXtMemcpy() - Copy the data from host to device
	result = hipfftXtMemcpy(planComplex, d_f, f, HIPFFT_COPY_HOST_TO_DEVICE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMemcpy failed\n");
		exit(EXIT_FAILURE);
	}

	// hipfftXtExecDescriptorC2C() - Execute FFT on data on multiple GPUs
	printf("Forward 2d FFT on multiple GPUs\n");
	result = hipfftXtExecDescriptorC2C(planComplex, d_f, d_f, HIPFFT_FORWARD);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtExecC2C  failed\n");
		exit(EXIT_FAILURE);
	}
//in this case reorder is not necessary , so next step not using d_d_f
	result = hipfftXtMemcpy(planComplex, d_d_f, d_f, HIPFFT_COPY_DEVICE_TO_DEVICE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMemcpy failed\n");
		exit(EXIT_FAILURE);
	}


	// hipfftXtMemcpy() - Copy data from multiple GPUs to host
	result = hipfftXtMemcpy(planComplex, h_d_out, d_d_f, HIPFFT_COPY_DEVICE_TO_HOST);

	if (result != HIPFFT_SUCCESS) {
		printf("*XtMemcpy failed\n");
		exit(EXIT_FAILURE);
	}


	// cleanup memory

	free(h_d_out);
	free(worksize);
    printf("  GPU Processing time: %f (ms)\n\n", GetTimer());

	return 0;
}
