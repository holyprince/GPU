/**
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/


// System includes
#include <stdlib.h>
#include <stdio.h>

#include <string.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

//CUFFT Header file
#include <hipfft/hipfftXt.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>

// Complex data type
typedef float2 Complex;


#define N 32

// Forward Declaration
__global__ void vectorAdd(hipfftComplex *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
    	A[i].y = i;
        A[i].x = A[i].x+(i)*10;
    }
}


__global__ void vectorAddself(hipfftComplex *A, int numElements, int curnum)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int newrow= i / (N/2);
    int newcol= i % (N/2);
    int realindex;
    if (i < numElements)
    {
    	realindex = newrow * N + newcol + (curnum *N/2);
    	A[i].y = realindex;
    	A[i].x = A[i].x + realindex*10;
    }


}
void vectorAdd2(hipLibXtDesc *d_ft, int nGPUs)
{
    int device ;
	int threadsPerBlock = 256;
	int blocksPerGrid =(N/2*N + threadsPerBlock - 1) / threadsPerBlock;
    for(int i=0; i < nGPUs ; i++)
    {
        device = d_ft->descriptor->GPUs[i];
        hipSetDevice(device) ;
        vectorAddself<<<blocksPerGrid,threadsPerBlock>>>((hipfftComplex*) d_ft->descriptor->data[i], N/2*N, i);

    }

    // Wait for device to finish all operation
    for(int i=0; i< nGPUs ; i++)
    {
        device = d_ft->descriptor->GPUs[i];
        hipSetDevice( device );
        hipDeviceSynchronize();
    }

}


///////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{

	int threadsPerBlock;
	int blocksPerGrid ;
	int GPU_N;
	(hipGetDeviceCount(&GPU_N));
	int nGPUs = 2;
	int *whichGPUs;
	whichGPUs = (int*) malloc(sizeof(int) * nGPUs);

	Complex *f = (Complex*) malloc(sizeof(Complex) * N * N);
	float *u_a = (float*) malloc(sizeof(float) * N * N);
	Complex *h_d_out = (Complex *) malloc(sizeof(Complex) * N * N);

	for (int i = 0; i < N * N; i++) {
		f[i].x = i % 5000 ;
		f[i].y = 0;
	}

	// cufftCreate() - Create an empty plan
	hipfftResult result;
	hipfftHandle planComplex;
	result = hipfftCreate(&planComplex);
	if (result != HIPFFT_SUCCESS) {
		printf("hipfftCreate failed\n");
		exit(EXIT_FAILURE);
	}

	// cufftXtSetGPUs() - Define which GPUs to use
	result = hipfftXtSetGPUs(planComplex, nGPUs, whichGPUs);

	if (result == HIPFFT_INVALID_DEVICE) {
		printf("This sample requires two GPUs on the same board.\n");
		printf("No such board was found. Waiving sample.\n");

	} else if (result != HIPFFT_SUCCESS) {
		printf("hipfftXtSetGPUs failed\n");

	}

	//Print the device information to run the code
	printf("\nRunning on GPUs\n");

	size_t* worksize;
	worksize = (size_t*) malloc(sizeof(size_t) * nGPUs);

	// cufftMakePlan2d() - Create the plan
	result = hipfftMakePlan2d(planComplex, N, N, HIPFFT_C2C, worksize);

	if (result != HIPFFT_SUCCESS) {
		printf("*MakePlan* failed\n");
		exit(EXIT_FAILURE);
	}

	// Create a variable on device
	// d_f - variable on device to store the input data
	// d_d_f - variable that store the natural order of d_f data
	// d_out - device output
	hipLibXtDesc *d_f, *d_d_f;

	result = hipfftXtMalloc(planComplex, (hipLibXtDesc **) &d_f, HIPFFT_XT_FORMAT_INPLACE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMalloc failed\n");
		exit(EXIT_FAILURE);
	}
	result = hipfftXtMalloc(planComplex, (hipLibXtDesc **) &d_d_f, HIPFFT_XT_FORMAT_INPLACE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMalloc failed\n");
		exit(EXIT_FAILURE);
	}

	// cufftXtMemcpy() - Copy the data from host to device
	result = hipfftXtMemcpy(planComplex, d_f, f, HIPFFT_COPY_HOST_TO_DEVICE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMemcpy failed\n");
		exit(EXIT_FAILURE);
	}

	// cufftXtExecDescriptorC2C() - Execute FFT on data on multiple GPUs
	printf("Forward 2d FFT on multiple GPUs\n");
	result = hipfftXtExecDescriptorC2C(planComplex, d_f, d_f, HIPFFT_FORWARD);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtExecC2C  failed\n");
		exit(EXIT_FAILURE);
	}
//in this case reorder is not necessary , so next step not using d_d_f
	result = hipfftXtMemcpy(planComplex, d_d_f, d_f, HIPFFT_COPY_DEVICE_TO_DEVICE);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtMemcpy failed\n");
		exit(EXIT_FAILURE);
	}

	vectorAdd2(d_f,nGPUs);


	// cufftXtMemcpy() - Copy data from multiple GPUs to host
	result = hipfftXtMemcpy(planComplex, h_d_out, d_f, HIPFFT_COPY_DEVICE_TO_HOST);

	if (result != HIPFFT_SUCCESS) {
		printf("*XtMemcpy failed\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < N*N ; i++) {
		if(i%32==0)
			printf("\n");
		printf("%f %f ", h_d_out[i].x, h_d_out[i].y);
	}

	// cleanup memory

	free(h_d_out);
	free(worksize);

	result = hipfftXtFree(d_f);
	if (result != HIPFFT_SUCCESS) {
		printf("*XtFree failed\n");
		exit(EXIT_FAILURE);
	}
	hipfftXtFree(d_d_f);
	// cufftDestroy() - Destroy FFT plan
	result = hipfftDestroy(planComplex);
	if (result != HIPFFT_SUCCESS) {
		printf("hipfftDestroy failed: code %d\n", (int) result);
		exit(EXIT_FAILURE);
	}
	printf(" single gpu for fft \n \n \n");

	int Ndim[2];
	Ndim[0] = N, Ndim[1] = N;
	int LENGTH = Ndim[0] * Ndim[1];

	hipfftComplex *output = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));

	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, Ndim[0] * Ndim[1] * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, Ndim[0] * Ndim[1] * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, f, Ndim[0] * Ndim[1] * sizeof(hipfftComplex), hipMemcpyHostToDevice);

	hipfftHandle plan;

	hipfftPlan2d(&plan, Ndim[0], Ndim[1], HIPFFT_C2C);

	hipfftExecC2C(plan, d_inputData, d_inputData, HIPFFT_FORWARD);

	threadsPerBlock = 256;
    blocksPerGrid =(N*N + threadsPerBlock - 1) / threadsPerBlock;
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_inputData, N*N);
	hipMemcpy(output, d_inputData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < N*N; i++) {
		if(i%32==0)
			printf("\n");
			printf("%f %f ", output[i].x, output[i].y);
	}

	hipfftDestroy(plan);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);
	return 0;
}



