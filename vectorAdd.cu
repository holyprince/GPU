#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"   //   /usr/local/cuda-9.0/include
#include "complex.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAdd(int *A, int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        //C[i].x = A[i] * B[i];
        C[i] = A[i]+B[i];
    }

    //printf("%f from gpu \n",C[i]);

}

int main()
{
	int *a,*b;
	int N=2000;
	a = (int *)malloc(sizeof(int)*N);
	b = (int *)malloc(sizeof(int)*N);
	for(int i=0;i<N;i++)
	{
		a[i]=1;
		b[i]=2;
	}
	int *c;
	c = (int *)malloc(sizeof(int)*N);
	int *d_a,*d_b,*d_c;
	hipMalloc((void**) &d_a, N * sizeof(int));
	hipMalloc((void**) &d_b, N * sizeof(int));
	hipMemcpy(d_a, a, N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N * sizeof(int),hipMemcpyHostToDevice);
	hipMalloc((void**) &d_c, N * sizeof(int));
    int threadsPerBlock = 256;
    int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
	hipMemcpy(c, d_c, N * sizeof(int),hipMemcpyDeviceToHost);    
	for(int i=0;i<10;i++)
		printf("%d \n",c[i]);
}