#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "time.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#define STREAMNUM 2
void init()
{

}
__global__ void MyKernel(float *a,float* b,int nx)
{

}

//#define PINDIRECT
int main() {

	int size=sizeof(float)*100;
	hipStream_t stream[2];
	for (int i = 0; i < 2; ++i)
		hipStreamCreate(&stream[i]);
	float* hostPtr;
	float* inputDevPtr,*outputDevPtr;
#ifdef PINDIRECT
    cudaMallocHost(&hostPtr, 2 * size);
#else
    hostPtr = (float*)malloc(2 * size);
    hipHostRegister(hostPtr,2 * size,hipHostMallocMapped);
#endif
    hipMalloc((void**)&inputDevPtr,size*STREAMNUM);
    hipMalloc((void**)&outputDevPtr,size*STREAMNUM);

    init();

	for (int i = 0; i < 2; i++) {

		hipMemcpyAsync(inputDevPtr + i * size, hostPtr + i * size, size,hipMemcpyHostToDevice, stream[i]);

		MyKernel<<<100, 512, 0, stream[i]>>>(outputDevPtr + i * size,inputDevPtr + i * size, size);

		hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size, size,hipMemcpyDeviceToHost, stream[i]);
	}

	for (int i = 0; i < 2; i++) {
		hipStreamSynchronize(stream[i]);
		hipStreamDestroy(stream[i]);
		//CPU code can execute cpu and gpu the same time ;
	}
}
