#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <assert.h>

typedef half2 ftype;

#ifndef PI
#define PI 3.14159265358979323846
#endif
#define Ndim 16
#define DataRange 50
#define PADDIM 64



void testarithmetic()
{
	ftype *d_adata;
	ftype *acpu_data=(ftype* )malloc(sizeof(ftype) * 2);
	hipMalloc(&d_adata, sizeof(ftype) * 2);
	hipMemset(d_adata, 0, 2 * sizeof(ftype));


    int threadsPerBlock = 256;
    int blocksPerGrid =(2 + threadsPerBlock - 1) / threadsPerBlock;
	vectorMulti_multitest<<<blocksPerGrid, threadsPerBlock>>>(d_adata, 2);
	hipMemcpy(acpu_data, d_adata, 2 * sizeof(ftype),hipMemcpyDeviceToHost);
	printf("%f %f \n",(float)acpu_data[0].x,(float)acpu_data[0].y);
}
int main(){

	testarithmetic();
	return 0;
}