
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <assert.h>

typedef half2 ftype;

#ifndef PI
#define PI 3.14159265358979323846
#endif
#define Ndim 16
#define DataRange 50
#define PADDIM 64

void fft1d(int dimx)
{

	int N[1];
	N[0] = dimx;
	int LENGTH = N[0] ;
	hipfftComplex *input = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i].x = (i % DataRange) /100.0;
		input[i].y = 0;
	}

	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, LENGTH * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipfftHandle plan;
	hipfftPlan1d(&plan, N[0], HIPFFT_C2C,1);
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	for (int i = 0; i < 1; i++) {

		int res=hipfftExecC2C(plan, d_inputData, d_outData, HIPFFT_FORWARD);
		printf("%d ",res);
	}
	size_t workSize;
	hipfftGetSize1d(plan, N[0], HIPFFT_C2C, 1, &workSize);
	printf("worksize : %ld and complex size %ld \n", workSize, LENGTH * sizeof(hipfftComplex));

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	hipMemcpy(output, d_outData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);


	printf("=======fft\n");
	for(int i=0;i<10;i++)
		printf("%f %f\n",output[i].x,output[i].y);

	printf("Time is %f \n",msecTotal1);
	hipfftDestroy(plan);
	free(input);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);

}

//OK
void testfp16fft(int dimx)
{

	int LENGTH= dimx;
	long long sig_size = LENGTH;

	hipfftComplex *bn=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH);
	ftype *input=(ftype*)malloc(sizeof(ftype)*LENGTH);

	ftype *acpu_data=(ftype*)malloc(sizeof(ftype)*LENGTH);
	ftype *bcpu_data=(ftype*)malloc(sizeof(ftype)*LENGTH);

	int i;
	for (i = 0; i < LENGTH; i++) {
		float temp;
		temp=i;
		acpu_data[i].x = temp;
		acpu_data[i].y = temp-temp;
	}

	for(int i=0;i<10;i++)
		printf("%f %f\n",(float)acpu_data[i].x,(float)acpu_data[i].y);

	ftype *d_adata;
	ftype *d_bdata;
	hipMalloc(&d_adata, sizeof(ftype) * sig_size);
	hipMalloc(&d_bdata, sizeof(ftype) * sig_size);


	hipMemset(d_adata, 0, sig_size * sizeof(ftype));
	hipMemset(d_bdata, 0, sig_size * sizeof(ftype));

	hipMemcpy(d_adata, acpu_data, LENGTH * sizeof(ftype),hipMemcpyHostToDevice);
	hipMemcpy(d_bdata, bcpu_data, LENGTH * sizeof(ftype),hipMemcpyHostToDevice);




	ftype *h_idata = (ftype *) malloc(sig_size * sizeof(ftype));

	hipfftHandle plan;
	hipfftResult r;
	r = hipfftCreate(&plan);
	size_t ws = 0;
	r = hipfftXtMakePlanMany(plan, 1, &sig_size, NULL, 1, 1, HIP_C_16F, NULL, 1,
			1, HIP_C_16F, 1, &ws, HIP_C_16F);

/*	assert(r == CUFFT_SUCCESS);
	r = cufftXtExec(plan, d_idata, d_odata, CUFFT_FORWARD); // warm-up
	assert(r == CUFFT_SUCCESS);
	cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);
	cudaEventRecord(start);*/


	hipfftXtExec(plan, d_adata, d_adata, HIPFFT_FORWARD);
	hipfftXtExec(plan, d_bdata, d_bdata, HIPFFT_FORWARD);
/*	assert(r == CUFFT_SUCCESS);
	cudaEventRecord(stop);
	cudaEventSynchronize(stop);
	float et;
	cudaEventElapsedTime(&et, start, stop);
	printf("forward FFT time for %ld samples: %fms\n", sig_size, et);*/

	hipMemcpy(input, d_adata, LENGTH * sizeof(ftype),hipMemcpyDeviceToHost);



}
int main(){

	fft1d(Ndim);
	testfp16fft(Ndim);

	return 0;
}
