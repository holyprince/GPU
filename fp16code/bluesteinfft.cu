
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <assert.h>

typedef half2 ftype;

#ifndef PI
#define PI 3.14159265358979323846
#endif
#define Ndim 723
#define DataRange 500
#define PADDIM 4096


void initdata(hipfftComplex *input,int LENGTH)
{
	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i].x = (i % DataRange);
		input[i].y = 0;
	}

}

void fft1d(int dimx)
{

	int N[1];
	N[0] = dimx;
	int LENGTH = N[0] ;
	hipfftComplex *input = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	initdata(input,LENGTH);

	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, LENGTH * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipfftHandle plan;
	hipfftPlan1d(&plan, N[0], HIPFFT_C2C,1);
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	for (int i = 0; i < 1; i++) {

		int res=hipfftExecC2C(plan, d_inputData, d_outData, HIPFFT_FORWARD);

	}
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("time is : %f \n",msecTotal1);


	size_t workSize;
	hipfftGetSize1d(plan, N[0], HIPFFT_C2C, 1, &workSize);
	printf("worksize : %ld and complex size %ld \n", workSize, LENGTH * sizeof(hipfftComplex));



	hipMemcpy(output, d_outData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);


	printf("=======fft\n");
	for(int i=0;i<10;i++)
		printf("%f %f\n",output[i].x,output[i].y);

	printf("Time is %f \n",msecTotal1);
	hipfftDestroy(plan);
	free(input);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);

}

__global__ void vectorMulti(hipfftComplex *A, hipfftComplex *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
    	hipfftComplex temp;
    	temp.x = A[i].x*B[i].x - A[i].y*B[i].y;
    	temp.y = A[i].x*B[i].y + A[i].y*B[i].x;
    	A[i].x=temp.x; A[i].y=temp.y;
    	//A[i]=cuCmulf(A[i],B[i]);
    }

}
__global__ void vectorMultianddiv(hipfftComplex *A, hipfftComplex *B, int numElements,int dimx)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
    	hipfftComplex temp;
    	temp.x = A[i].x*B[i].x - A[i].y*B[i].y;
    	temp.y = A[i].x*B[i].y + A[i].y*B[i].x;
    	A[i].x=temp.x/dimx; A[i].y=temp.y/dimx;

    	//A[i]=cuCmulf(A[i],B[i]);
    }

}

__global__ void vectorMulti_fp16(half2 *A, half2 *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
       	hipfftComplex temp;
        temp.x = reinterpret_cast<half&>(A[i].x)*reinterpret_cast<half&>(B[i].x) - reinterpret_cast<half&>(A[i].y)*reinterpret_cast<half&>(B[i].y);
        temp.y = reinterpret_cast<half&>(A[i].x)*reinterpret_cast<half&>(B[i].y) + reinterpret_cast<half&>(A[i].y)*reinterpret_cast<half&>(B[i].x);
        reinterpret_cast<half&>(A[i].x)=temp.x; reinterpret_cast<half&>(A[i].y)=temp.y;
    	//A[i]=__hmul2 (A[i],B[i]);
    }

}

__global__ void vectordivide(hipfftComplex *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
       	A[i].x=A[i].x/numElements; A[i].y=A[i].y/numElements;

    }

}

__global__ void vectordivide_fp16(half2 *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
       	reinterpret_cast<half&>(A[i].x)= (reinterpret_cast<half&>(A[i].x)/(half)numElements);
     	reinterpret_cast<half&>(A[i].y)= (reinterpret_cast<half&>(A[i].y)/(half)numElements);
    }

}


__global__ void vectorMultibkstar(hipfftComplex *A, int numElements,int divdata,int flag)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {

    	int index=i+divdata;

    	hipfftComplex temp;
    	temp.x=cos((PI*i*i)/divdata);
    	temp.y=-sin((PI*i*i)/divdata)*flag;

       	hipfftComplex temp2;
        temp2.x = A[index].x*temp.x - A[index].y*temp.y;
        temp2.y = A[index].x*temp.y + A[index].y*temp.x;
       	A[index].x=temp2.x; A[index].y=temp2.y;

    	//A[i]=cuCmulf(temp,A[i]);
    }

}

__global__ void vectorMultibkstar2(hipfftComplex *A, int numElements,int divdata)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {

    	int index=i+divdata;

    	hipfftComplex temp;
    	temp.x=cos((PI*i*i)/divdata);
    	temp.y=sin((PI*i*i)/divdata);

       	hipfftComplex temp2;
        temp2.x = A[index].x*temp.x - A[index].y*temp.y;
        temp2.y = A[index].x*temp.y + A[index].y*temp.x;
       	A[index].x=temp2.x; A[index].y=temp2.y;

    	//A[i]=cuCmulf(temp,A[i]);
    }

}
__global__ void vectorMultibkstar_fp16(half2 *A, int numElements,int divdata)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {

    	int index=i+divdata;

    	half2 temp;
    	reinterpret_cast<half&>(temp.x)=cos((PI*i*i)/divdata);
    	reinterpret_cast<half&>(temp.y)=-sin((PI*i*i)/divdata);
 //   	A[index]=__hmul2(A[index],temp);

     	half2 temp2;
        reinterpret_cast<half&>(temp2.x) = reinterpret_cast<half&>(A[index].x)*(half)reinterpret_cast<half&>(temp.x) - reinterpret_cast<half&>(A[index].y)*reinterpret_cast<half&>(temp.y);
        reinterpret_cast<half&>(temp2.y) = reinterpret_cast<half&>(A[index].x)*(half)reinterpret_cast<half&>(temp.y) + reinterpret_cast<half&>(A[index].y)*reinterpret_cast<half&>(temp.x);
       	reinterpret_cast<half&>(A[index].x)=reinterpret_cast<half&>(temp2.x); reinterpret_cast<half&>(A[index].y)=reinterpret_cast<half&>(temp2.y);
    	//A[i]=cuCmulf(temp,A[i]);
    }

}



void bluestein(int dimx)
{
	//step1:
	int padx=PADDIM;
	int LENGTH= dimx;
	hipfftComplex *an=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH);
	hipfftComplex *bn=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH*2);
	hipfftComplex *input=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH);

	initdata(input,LENGTH);
	for(int i=0;i<LENGTH;i++)
	{
		hipfftComplex temp;
		temp.x=cos((PI*i*i)/LENGTH);
		temp.y=sin((PI*i*i)/LENGTH);
		an[i].x=input[i].x*temp.x - input[i].y*temp.y ;
		an[i].y=-(input[i].x*temp.y + input[i].y*temp.x) ;
	}
	for(int i=0;i<LENGTH*2;i++)
	{
		bn[i].x=cos((PI*(i-LENGTH)*(i-LENGTH))/LENGTH);
		bn[i].y=sin((PI*(i-LENGTH)*(i-LENGTH))/LENGTH);
	}


	//convlution
	hipfftComplex *d_aData,*d_bData;

	hipfftHandle plan;
	hipfftPlan1d(&plan, padx, HIPFFT_C2C,1);
	hipMalloc((void**) &d_aData, padx * sizeof(hipfftComplex));
	hipMalloc((void**) &d_bData, padx * sizeof(hipfftComplex));


	hipMemset(d_aData, 0, padx * sizeof(hipfftComplex));
	hipMemset(d_bData, 0, padx * sizeof(hipfftComplex));


	hipMemcpy(d_aData, an, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipMemcpy(d_bData, bn, LENGTH*2 * sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipfftExecC2C(plan, d_aData, d_aData, HIPFFT_FORWARD);


	hipfftExecC2C(plan, d_bData, d_bData, HIPFFT_FORWARD);

    int threadsPerBlock = 256;
    int blocksPerGrid =(padx + threadsPerBlock - 1) / threadsPerBlock;
    vectorMulti<<<blocksPerGrid, threadsPerBlock>>>(d_aData, d_bData, padx);

	hipMemcpy(input, d_aData, LENGTH * sizeof(hipfftComplex),hipMemcpyDeviceToHost);


    //cudaDeviceSynchronize();
	hipfftExecC2C(plan, d_aData, d_aData, HIPFFT_BACKWARD);

	vectordivide<<<blocksPerGrid, threadsPerBlock>>>(d_aData, padx);
	vectorMultibkstar<<<blocksPerGrid, threadsPerBlock>>>(d_aData, padx,dimx,1);


	hipMemcpy(input, d_aData+ dimx, LENGTH * sizeof(hipfftComplex),hipMemcpyDeviceToHost);

	for(int i=0;i<10;i++)
		printf("%f %f\n",input[i].x,input[i].y);

/*
 * inverse test
	for(int i=0;i<LENGTH;i++)
	{
		cufftComplex temp;
		temp.x=cos((PI*i*i)/LENGTH);
		temp.y=sin((PI*i*i)/LENGTH);
		an[i].x=input[i].x*temp.x - input[i].y*temp.y ;
		an[i].y=input[i].x*temp.y + input[i].y*temp.x ;
	}


	for(int i=0;i<LENGTH*2;i++)
	{
		bn[i].x=cos((PI*(i-LENGTH)*(i-LENGTH))/LENGTH);
		bn[i].y=-sin((PI*(i-LENGTH)*(i-LENGTH))/LENGTH);
	}
	cudaMemset(d_aData, 0, padx * sizeof(cufftComplex));
	cudaMemset(d_bData, 0, padx * sizeof(cufftComplex));

	cudaMemcpy(d_aData, an, LENGTH * sizeof(cufftComplex),cudaMemcpyHostToDevice);
	cudaMemcpy(d_bData, bn, LENGTH*2 * sizeof(cufftComplex),cudaMemcpyHostToDevice);
	cufftExecC2C(plan, d_aData, d_aData, CUFFT_FORWARD);


	cudaMemcpy(input, d_aData , LENGTH * sizeof(cufftComplex),cudaMemcpyDeviceToHost);



	cufftExecC2C(plan, d_bData, d_bData, CUFFT_FORWARD);

    threadsPerBlock = 256;
    blocksPerGrid =(padx + threadsPerBlock - 1) / threadsPerBlock;
    vectorMultianddiv<<<blocksPerGrid, threadsPerBlock>>>(d_aData, d_bData, padx,dimx);

    //cudaDeviceSynchronize();
	cufftExecC2C(plan, d_aData, d_aData, CUFFT_INVERSE);
	vectordivide<<<blocksPerGrid, threadsPerBlock>>>(d_aData, padx);

	vectorMultibkstar<<<blocksPerGrid, threadsPerBlock>>>(d_aData, padx,dimx,-1);

	cudaMemcpy(input, d_aData + dimx, LENGTH * sizeof(cufftComplex),cudaMemcpyDeviceToHost);
	printf("\n");
	for(int i=0;i<10;i++)
		printf("%f %f\n",input[i].x,input[i].y);
*/

}

void bluesteininverse(int dimx)
{
	int padx=PADDIM;
	int LENGTH= dimx;
	hipfftComplex *an=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH);
	hipfftComplex *bn=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH*2);
	hipfftComplex *input=(hipfftComplex*)malloc(sizeof(hipfftComplex)*LENGTH);

	initdata(input,LENGTH);


	for(int i=0;i<LENGTH;i++)
	{
		hipfftComplex temp;
		temp.x=cos((PI*i*i)/LENGTH);
		temp.y=sin((PI*i*i)/LENGTH);
		an[i].x=input[i].x*temp.x - input[i].y*temp.y ;
		an[i].y=input[i].x*temp.y + input[i].y*temp.x ;
	}

	for(int i=0;i<LENGTH*2;i++)
	{
		bn[i].x=cos((PI*(i-LENGTH)*(i-LENGTH))/LENGTH);
		bn[i].y=-sin((PI*(i-LENGTH)*(i-LENGTH))/LENGTH);
	}


	//convlution
	hipfftComplex *d_aData,*d_bData;

	hipfftHandle plan;
	hipfftPlan1d(&plan, padx, HIPFFT_C2C,1);
	hipMalloc((void**) &d_aData, padx * sizeof(hipfftComplex));
	hipMalloc((void**) &d_bData, padx * sizeof(hipfftComplex));


	hipMemset(d_aData, 0, padx * sizeof(hipfftComplex));
	hipMemset(d_bData, 0, padx * sizeof(hipfftComplex));


	hipMemcpy(d_aData, an, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipMemcpy(d_bData, bn, LENGTH*2 * sizeof(hipfftComplex),hipMemcpyHostToDevice);
	hipfftExecC2C(plan, d_aData, d_aData, HIPFFT_FORWARD);


	hipfftExecC2C(plan, d_bData, d_bData, HIPFFT_FORWARD);

    int threadsPerBlock = 256;
    int blocksPerGrid =(padx + threadsPerBlock - 1) / threadsPerBlock;
    vectorMultianddiv<<<blocksPerGrid, threadsPerBlock>>>(d_aData, d_bData, padx,dimx);



	hipfftExecC2C(plan, d_aData, d_aData, HIPFFT_BACKWARD);
	vectordivide<<<blocksPerGrid, threadsPerBlock>>>(d_aData, padx);

	vectorMultibkstar<<<blocksPerGrid, threadsPerBlock>>>(d_aData, padx,dimx,-1);

	hipMemcpy(input, d_aData + dimx, LENGTH * sizeof(hipfftComplex),hipMemcpyDeviceToHost);
	printf("\n");
	for(int i=0;i<10;i++)
		printf("%f %f\n",input[i].x,input[i].y);

}






int main(){

	fft1d(Ndim);
	bluestein(Ndim);
	//fft1d(4096);
	printf("FP16: \n");




	return 0;
}


