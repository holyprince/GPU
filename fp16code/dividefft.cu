#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

#include "time.h"



void testmoduleGPU(int dimx,int dimy,int dimz) {
	int N[3];
	N[0] = dimx; N[1] = dimy; N[2] = dimz;
	int LENGTH = N[0] * N[1] * N[2];
	hipfftComplex *input = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*) malloc(
			LENGTH * sizeof(hipfftComplex));
	int i;
	for (i = 0; i < LENGTH; i++) {
		input[i].x = i % 1000;
		input[i].y = 0;
	}

	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, LENGTH * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, input, LENGTH * sizeof(hipfftComplex),
			hipMemcpyHostToDevice);

	hipfftHandle plan;

	int t= hipfftPlan3d(&plan, N[0], N[1], N[2], HIPFFT_C2C);

	hipfftExecC2C(plan, d_inputData, d_outData, HIPFFT_FORWARD);

	hipMemcpy(output, d_outData, LENGTH * sizeof(hipfftComplex),
			hipMemcpyDeviceToHost);

	for (i = 0; i < 10; i++) {
		printf("%f %f \n", output[i].x, output[i].y);
	}
	for(int i=0+dimx*dimy;i<10+dimx*dimy;i++)
		printf("%f %f \n",output[i].x,output[i].y);
	hipfftDestroy(plan);
	free(input);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);
}


void dividefft(int dimx,int dimy,int dimz)
{
	int N[3];
	N[0] = dimx; N[1] = dimy; N[2] = dimz;
	int LENGTH = N[0] * N[1] * N[2];
	hipfftComplex *cinput = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));

	for(int i=0;i<LENGTH;i++)
	{
		cinput[i].x=i% 1000;
		cinput[i].y=0;
	}

	hipfftComplex *d_inputData;
	hipMalloc((void**) &d_inputData, LENGTH * sizeof(hipfftComplex));

	hipMemcpy(d_inputData, cinput, LENGTH * sizeof(hipfftComplex),hipMemcpyHostToDevice);

	hipfftHandle plan;

	// for one dimension

	int BATCH= dimy * dimz;
	hipfftPlan1d(&plan, dimx , HIPFFT_C2C, BATCH);
	hipfftExecC2C(plan, d_inputData, d_inputData,HIPFFT_FORWARD);
	hipfftDestroy(plan);


	// for two dimension
	int dimn[1];
	dimn[0]=dimy;
	int inembed[2];
	inembed[0]=dimx;inembed[1]=dimy;
	int istride = dimx;
	int idist=1;
	int onembed[2];
	onembed[0]=dimx;onembed[1]=dimy;
	int ostride = dimx;
	int odist=1;
	hipfftPlanMany(&plan, 1, dimn, inembed,istride, idist, onembed, ostride,odist,HIPFFT_C2C, dimx);

	for(int i=0;i<dimz;i++)
		hipfftExecC2C(plan, d_inputData+ i*(dimx*dimy), d_inputData + i*(dimx*dimy), HIPFFT_FORWARD);
	hipfftDestroy(plan);


	// for three dimension
	int dimnz[0];
	dimnz[0]=dimz;
	int inembedz[3],outembedz[3];
	inembedz[0]=dimx;inembedz[1]=dimy;inembedz[2]=dimz;
	int istridez = dimx*dimy;
	int idistz=1;
	outembedz[0]=dimx;outembedz[1]=dimy;outembedz[2]=dimz;
	int ostridez = dimx*dimy;
	int odistz=1;


	hipfftPlanMany(&plan, 1, dimnz, inembedz,istridez, idistz, outembedz, ostridez,odistz, HIPFFT_C2C, dimx*dimy);
	hipfftExecC2C(plan, d_inputData, d_inputData, HIPFFT_FORWARD);




	hipMemcpy(cinput, d_inputData , LENGTH * sizeof(hipfftComplex),hipMemcpyDeviceToHost);


	for (int i = 0; i < 10; i++) {
		printf("%f %f \n", cinput[i].x, cinput[i].y);
	}
	for(int i=0+dimx*dimy;i<10+dimx*dimy;i++)
		printf("%f %f \n",cinput[i].x,cinput[i].y);

	hipfftDestroy(plan);
	hipFree(cinput);


}

void dividefft2d1d(int NX,int NY,int NZ)
{
    int res;
    int LENGTH= NX*NY*NZ;
    int dimx=NX; int dimy=NY; int dimz=NZ;
    hipfftComplex *d_in ;

	hipfftComplex *cinput = (hipfftComplex*) malloc(LENGTH * sizeof(hipfftComplex));

	for(int i=0;i<LENGTH;i++)
	{
		cinput[i].x=i% 1000;
		cinput[i].y=0;
	}
    hipMalloc((void**) & (d_in), sizeof(hipfftComplex) * NX*NY*NZ);
    hipMemcpy(d_in, cinput, NX*NY*NZ * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    size_t worksize=0;
    int xyN[2];
    xyN[0]=NX;
    xyN[1]=NY;
    hipfftHandle xyplan;
    res=hipfftPlanMany(&xyplan, 2, xyN, NULL, 0, 0, NULL, 0, 0, HIPFFT_C2C, NZ);
    //printf("check1: %d\n",res);
    res=hipfftExecC2C(xyplan, d_in, d_in, HIPFFT_FORWARD);
    //printf("check2: %d\n",res);
    res=hipfftGetSizeMany(xyplan,2, xyN, NULL, 0, 0, NULL, 0, 0, HIPFFT_C2C, NZ, &worksize);
    //printf(" worksize 1 %ld \n ",worksize);
    hipfftDestroy(xyplan);
    int zN[1];
    zN[0]=NZ;
    int inembed[3];
    inembed[0]=NX;inembed[1]=NY;inembed[2]=NZ;

    hipfftHandle zplan;
    res=hipfftPlanMany(&zplan, 1, zN, inembed, NX*NY, 1, inembed, NX*NY, 1, HIPFFT_C2C, NX*NY);
    //printf("check3: %d\n",res);
    res=hipfftGetSizeMany(zplan,1,zN, inembed, NX*NY, 1, inembed, NX*NY, 1, HIPFFT_C2C,NX*NY,&worksize);
    //printf(" worksize 2 %ld \n ",worksize);

    res=hipfftExecC2C(zplan, d_in, d_in, HIPFFT_FORWARD);
    //printf("check4: %d\n",res);
    hipMemcpy(cinput, d_in, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++) {
		printf("%f %f \n", cinput[i].x, cinput[i].y);
	}
	for(int i=0+dimx*dimy;i<10+dimx*dimy;i++)
		printf("%f %f \n",cinput[i].x,cinput[i].y);

    hipFree(d_in);
    hipfftDestroy(zplan);
}


int main() {
	//readdataGPU(10,10,10);
	testmoduleGPU(100,100,100);
	printf("\n");
	dividefft(100,100,100);
	printf("\n");
	dividefft2d1d(100,100,100);
}


