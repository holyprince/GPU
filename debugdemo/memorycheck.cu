#include "stdio.h"
#include "stdlib.h"
#include "fftw3.h"
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"

int main() {

	int N[3];
	N[0]=723;N[1]=723;N[2]=723;

	int Nsum=N[0]*N[1]*N[2];
	double *c_real_data;
	c_real_data = (double*) malloc(sizeof(double)*Nsum);

	for(int i=0;i<Nsum;i++)
	{
		c_real_data[i] = i%700;
	}
	hipfftResult cr;
	hipfftHandle cufftForwrdHandle;
	cr =  hipfftPlan3d(&cufftForwrdHandle, N[0], N[1],N[2], HIPFFT_C2R);
	size_t workSize,freeMem,totalMem;
	hipfftGetSize(cufftForwrdHandle, &workSize);
	printf("%d and worksize : %ld \n ",cr,workSize);
	hipMemGetInfo(&freeMem, &totalMem);
	printf("%ld %ld \n",freeMem,totalMem);
}