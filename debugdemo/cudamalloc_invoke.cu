#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "complex.h"
#include <stdio.h>
#include <stdlib.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void vectorMulti(double *A, double *B, hipfftComplex *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i].x = A[i] * B[i];

    }

    //printf("%f from gpu \n",C[i].x);

}

void initgpu()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("GPU num for max %d \n",devCount);
	hipSetDevice(0);
}



double * gpusetdata_double(double *d_data,int N ,double *c_data)
{
	HANDLE_ERROR (hipMalloc((void**) &d_data, N * sizeof(double)));
	HANDLE_ERROR ( hipMemcpy(d_data, c_data, N * sizeof(double),hipMemcpyHostToDevice));
	return d_data;
}
void gpusetdata_double_void(double *d_data,int N ,double *c_data)
{
	HANDLE_ERROR (hipMalloc((void**) &d_data, N * sizeof(double)));
	HANDLE_ERROR ( hipMemcpy(d_data, c_data, N * sizeof(double),hipMemcpyHostToDevice));
}


void vector_Multi(double *data1, double *data2, hipfftComplex *res,int numElements)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
	vectorMulti<<<blocksPerGrid, threadsPerBlock>>>(data1, data2, res, numElements);
}

void cpugetdata(hipfftComplex *d_outData, hipfftComplex *c_outData, int N)
{
	HANDLE_ERROR ( hipMemcpy(c_outData, d_outData , N * sizeof(hipfftComplex),hipMemcpyDeviceToHost));
}
hipfftComplex* gpumallocdata(hipfftComplex *d_outData,int N)
{
	HANDLE_ERROR( hipMalloc((void**) &d_outData,  N * sizeof(hipfftComplex)));
	return d_outData;
}

void printdatatofile(hipfftComplex *data,int N)
{
	FILE *fp= fopen("data1.out","w+");
	for(int i=0;i< N ;i++)
	{
		fprintf(fp,"%f %f |",data[i].x,data[i].y);
		if(i%100==0)
			fprintf(fp,"\n");
	}
	fclose(fp);
}

void printdata(hipfftComplex *data,int N)
{

	for(int i=0;i< N ;i++)
	{
		printf("%f %f |",data[i].x,data[i].y);
		if(i%100==0)
			printf("\n");
	}

}


int main()
{

	initgpu();
	hipfftComplex *d_inputData, *d_outData;
	int nzyxdim=2000;
	double *d_Fnewweight;
	double *d_Fweight;
	double *c_data;

	double * Fnewweightdata, *Fweightdata;
	Fnewweightdata =(double*)malloc(nzyxdim*sizeof(double));
	Fweightdata =(double*)malloc(nzyxdim*sizeof(double));
	c_data= (double*)malloc(nzyxdim*sizeof(double));
	for(int i=0;i<nzyxdim;i++)
	{
		Fnewweightdata[i]=1;
		Fweightdata[i]=2;
	}
	d_Fnewweight = gpusetdata_double(d_Fnewweight,nzyxdim,Fnewweightdata);
	d_Fweight= gpusetdata_double(d_Fweight,nzyxdim,Fweightdata);


	int Fconvnum=nzyxdim;
	d_outData = gpumallocdata(d_outData,Fconvnum);
	printf("Fconvnum  : %d \n",Fconvnum);

	hipfftComplex *c_output = (hipfftComplex*) malloc(Fconvnum * sizeof(hipfftComplex));
	vector_Multi(d_Fnewweight,d_Fweight,d_outData,nzyxdim);
	cpugetdata(d_outData,c_output,nzyxdim);

	printdata(c_output,100);
	return 0;

}