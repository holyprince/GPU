
#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"







#define NX 256
#define NY 128
#define NRANK 2
#define BATCH 1



int main() {

	int N=4;
	hipfftReal *d_real_data,*c_real_data;
	c_real_data = (hipfftReal*) malloc(sizeof(hipfftReal)*4);
    hipMalloc((void**)&d_real_data, N * sizeof(hipfftReal));
	c_real_data[0]=1;
	c_real_data[1]=2;
	c_real_data[2]=3;
	c_real_data[3]=4;

	hipMemcpy(d_real_data,c_real_data,sizeof(hipfftReal)*4,hipMemcpyHostToDevice);

	hipfftComplex *d_comp_data,*c_comp_data;
	hipMalloc((void**)&d_comp_data, sizeof(hipfftComplex)*(N/2+1));
	c_comp_data = (hipfftComplex*) malloc(sizeof(hipfftComplex)*(N/2+1));

	hipfftHandle cufftForwrdHandle;
	hipfftPlan1d(&cufftForwrdHandle, N, HIPFFT_R2C, 1);
	hipfftExecR2C(cufftForwrdHandle, d_real_data, d_comp_data);
	hipDeviceSynchronize();
	hipMemcpy(c_comp_data,d_comp_data,sizeof(hipfftComplex)*(N/2+1),hipMemcpyDeviceToHost);
	hipfftDestroy(cufftForwrdHandle);

	for(int i=0;i<(N/2+1);i++)
		printf("%f %f \n",c_comp_data[i].x,c_comp_data[i].y);

// C2R

	hipfftHandle cufftInverseHandle;
	hipMemset(d_real_data,0,N*sizeof(hipfftReal));
	hipfftPlan1d(&cufftInverseHandle, N, HIPFFT_C2R, 1);
	hipfftExecC2R(cufftInverseHandle, d_comp_data,d_real_data);
	hipMemcpy(c_real_data,d_real_data,sizeof(hipfftReal)*N,hipMemcpyDeviceToHost);
	for(int i=0;i<N;i++)
		printf("%f ",c_real_data[i]);


}
/*
ref: https://blog.csdn.net/congwulong/article/details/7576012
10.000000 -0.000000
-2.000000 2.000000
-2.000000 0.000000
4.000001 7.999999 11.999999 16.000000
 */