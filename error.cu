__host__ ​ __device__ ​const char* hipGetErrorName ( hipError_t error )
__host__ ​ __device__ ​const char* hipGetErrorString ( hipError_t error )

    hipError_t err;
    err = hipMemcpy(p_d, p_h, sizeof(float)*1024, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy : %s\n", hipGetErrorString(hipGetLastError()));
        exit(EXIT_FAILURE);
    }
	
#define checkCudaErrors( a ) do { \
    if (hipSuccess != (a)) { \
    fprintf(stderr, "Cuda runtime error in line %d of file %s \
    : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
    exit(EXIT_FAILURE); \
    } \
    } while(0);
//因此，这里，我们就使用这个宏来分析runtime api是否调用正确了：
checkCudaErrors( hipMemcpy(p_d, p_h, sizeof(float)*1024, hipMemcpyHostToDevice) );


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}	



#ifdef DEBUG_HIP
#define DEBUG_HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define DEBUG_PRIVATE_ERROR(func, status) {  \
                       (status) = (func); \
                       DEBUG_HANDLE_ERROR(status); \
                   }
#else
#define DEBUG_HANDLE_ERROR( err ) (err) //Do nothing
#define DEBUG_PRIVATE_ERROR( err ) (err) //Do nothing
#endif

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define PRIVATE_ERROR(func, status) {  \
                       (status) = (func); \
                       HANDLE_ERROR(status); \
                   }

static void HandleError( hipError_t err, const char *file, int line )
{

    if (err != hipSuccess)
    {
    	fprintf(stderr, "ERROR: %s in %s at line %d (error-code %d)\n",
						hipGetErrorString( err ), file, line, err );
		fflush(stdout);
#ifdef DEBUG_HIP
		raise(SIGSEGV);
#else
		CRITICAL(ERRGPUKERN);
#endif
    }
}


