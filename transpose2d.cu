#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "timer.h"

#define NX 2048
#define NY 2048

#define K 16

void initdata(float data[])
{
		for(int yi=0; yi< NY;yi++)
			for(int xi=0; xi< NX;xi++)
			{
				data[yi*NX + xi] = yi*NX + xi;
			}

}
void printdata(float data[])
{



		for(int yi=0; yi< NY;yi++) {
			for(int xi=0; xi< NX;xi++)
			{
				printf("%.2f ",data[yi*NX + xi]);
			}
			printf("\n");
		}
}
void transpose_CPU(float in[], float out[]) {


		for(int yi=0; yi< NY;yi++)
			for(int xi=0; xi< NX;xi++)
				out[ xi*NX + yi] = in[ yi*NX + xi];
}

__global__ void transpose(float in[], float out[])
{
    unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
//    unsigned int k = blockIdx.z * blockDim.z;
    //unsigned int idx = i * N + j;
    if(i < NX && j < NY )
    {

    	out[ i *NX+ j] = in[j*NX+ i];
    	//out[k * NXY + j *NX+ i] = in[k * NXY + j*NX+ i];
    }
}


__global__ void transpose_parallel_per_element_tiled1(float in[], float out[])
{
  int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
  int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

  int x = threadIdx.x, y = threadIdx.y;

  __shared__ float tile[K][K];

  tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * NX];
  __syncthreads();
  out[(out_corner_i + x) + (out_corner_j + y) * NX] = tile[x][y];
}


__global__ void transpose_parallel_per_element_tiled2(float in[], float out[])
{
  int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
  int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

  int x = threadIdx.x, y = threadIdx.y;

  __shared__ float tile[K][K+1];

  tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * NX];
  __syncthreads();
  out[(out_corner_i + x) + (out_corner_j + y) * NX] = tile[x][y];
}
#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void transpose_cudaapi(float in[], float out[],int width,int height)
{
	int xIndex = blockIdx.x *TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y *TILE_DIM + threadIdx.y;

	int index_in = xIndex+ width*yIndex;
	int index_out = yIndex+ height *xIndex;
	for(int i=0;i<TILE_DIM ; i+=BLOCK_ROWS){
		out[index_out +i]= in[index_in + i* width];
	}
}

__global__ void transpose_cudaapi_shared(float in[], float out[],int width,int height)
{
	__shared__ float tile[TILE_DIM][TILE_DIM+1];

	int xIndex = blockIdx.x *TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y *TILE_DIM + threadIdx.y;
	int index_in = xIndex+ width*yIndex;

	xIndex= blockIdx.y * TILE_DIM+ threadIdx.x;
	yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
	int index_out = xIndex+ yIndex*height;

	for(int i=0;i<TILE_DIM ; i+=BLOCK_ROWS){
		tile[threadIdx.y+i][threadIdx.x] =  in[index_in + i* width];
	}
	__syncthreads();

	for(int i=0;i<TILE_DIM ; i+=BLOCK_ROWS ){
		out[index_out + i*height] = tile[threadIdx.x][threadIdx.y+i];
	}
}


void testbandwidth()
{
	int devicenum=0;
    hipSetDevice(devicenum);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devicenum);
    printf("  Memory Clock rate:    %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:    %d-bit\n",   deviceProp.memoryBusWidth);
    printf(" Theory peak : %.2f GB/s \n",(deviceProp.memoryBusWidth*deviceProp.memoryClockRate * 1e-3f)/8/1000.0);
}
int main(int argc, char **argv) {


	testbandwidth();


	int numbytes =NX * NY * sizeof(float);
	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	initdata(in);

//	transpose_CPU(in,out);

	float *d_in, *d_out; // on Device
	hipMalloc((void **)&d_in, numbytes);
	hipMalloc((void **)&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);


	dim3 threads(K,K);
	dim3 blocks((NX + threads.x - 1) / threads.x, (NY + threads.y - 1) / threads.y);


	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
	transpose<<<blocks, threads>>>(d_in, d_out);
//	transpose_parallel_per_element_tiled<<<blocks, threads>>>(d_in, d_out);
//	hipDeviceSynchronize();
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time1  : %f \n",msecTotal1);
	}


	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	transpose_parallel_per_element_tiled1<<<blocks, threads>>>(d_in, d_out);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time2  : %f \n",msecTotal1);
	}

	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	transpose_parallel_per_element_tiled2<<<blocks, threads>>>(d_in, d_out);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time3  : %f \n",msecTotal1);
	}

	dim3 threads1(TILE_DIM, BLOCK_ROWS);
	dim3 blocks1(NX / TILE_DIM, NY / TILE_DIM);
	//dim3 blocks1((NX + TILE_DIM - 1) / TILE_DIM, (NY + TILE_DIM - 1) / TILE_DIM);

	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	transpose_cudaapi<<<blocks1, threads1>>>(d_in, d_out,NX,NY);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time4  : %f \n",msecTotal1);
	}
	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	transpose_cudaapi_shared<<<blocks1, threads1>>>(d_in, d_out,NX,NY);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time5  : %f \n",msecTotal1);
	}

	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);


//	printdata(out);
	return 0;
}
