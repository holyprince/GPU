#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "timer.h"

#define NX 1024
#define NY 1024

#define K 16

void initdata(float data[])
{
		for(int yi=0; yi< NY;yi++)
			for(int xi=0; xi< NX;xi++)
			{
				data[yi*NX + xi] = yi*NX + xi;
			}

}
void printdata(float data[])
{



		for(int yi=0; yi< NY;yi++) {
			for(int xi=0; xi< NX;xi++)
			{
				printf("%.2f ",data[yi*NX + xi]);
			}
			printf("\n");
		}
}
void transpose_CPU(float in[], float out[]) {


		for(int yi=0; yi< NY;yi++)
			for(int xi=0; xi< NX;xi++)
				out[ xi*NX + yi] = in[ yi*NX + xi];
}

__global__ void transpose(float in[], float out[])
{
    unsigned int i = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
//    unsigned int k = blockIdx.z * blockDim.z;
    //unsigned int idx = i * N + j;
    if(i < NX && j < NY )
    {

    	out[ i *NX+ j] = in[j*NX+ i];
    	//out[k * NXY + j *NX+ i] = in[k * NXY + j*NX+ i];
    }
}


__global__ void transpose_parallel_per_element_tiled(float in[], float out[])
{
  int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
  int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

  int x = threadIdx.x, y = threadIdx.y;

  __shared__ float tile[K][K+1];

  tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * NX];
  __syncthreads();
  out[(out_corner_i + x) + (out_corner_j + y) * NX] = tile[x][y];
}

void testbandwidth()
{
	int devicenum=0;
    hipSetDevice(devicenum);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devicenum);
    printf("  Memory Clock rate:    %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:    %d-bit\n",   deviceProp.memoryBusWidth);
    printf(" Theory peak : %.2f GB/s \n",(deviceProp.memoryBusWidth*deviceProp.memoryClockRate * 1e-3f)/8/1000.0);
}
int main(int argc, char **argv) {


	testbandwidth();


	int numbytes =NX * NY * sizeof(float);
	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	initdata(in);


//	transpose_CPU(in,out);

	float *d_in, *d_out; // on Device
	hipMalloc((void **)&d_in, numbytes);
	hipMalloc((void **)&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);


	dim3 threads(K,K);
	dim3 blocks((NX + threads.x - 1) / threads.x, (NY + threads.y - 1) / threads.y);


	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
	transpose<<<blocks, threads>>>(d_in, d_out);
//	transpose_parallel_per_element_tiled<<<blocks, threads>>>(d_in, d_out);
//	hipDeviceSynchronize();
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time1  : %f \n",msecTotal1);
	}


	for(int i=0;i<21;i++)
	{
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	transpose_parallel_per_element_tiled<<<blocks, threads>>>(d_in, d_out);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("Time2  : %f \n",msecTotal1);
	}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);


//	printdata(out);
	return 0;
}
